// EXAMPLE TAKEN FROM: https://leimao.github.io/blog/Proper-CUDA-Error-Checking/

#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char *const func, const char *const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
    else
    {
        std::cout << "CUDA runtime API call at: " << file << ":" << line
                  << " is ok!" << std::endl;
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char *const file, const int line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
    else
    {
        std::cout << "CUDA runtime API call at: " << file << ":" << line
                  << " is ok!" << std::endl;
    }
}

__global__ void bad_kernel(int i)
{
    assert(false);
}

int main()
{
    float *p;
    // This will produce error.
    CHECK_CUDA_ERROR(hipMalloc(&p, 10 * sizeof(float)));
    bad_kernel<<<1, 1>>>(10);
    CHECK_CUDA_ERROR(hipMalloc(&p, 10 * sizeof(float)));
}